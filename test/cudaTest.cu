
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void printThread(){
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  printf("ThreadIdx : %d \n",i);
  
}


int main(){

  size_t size = 16*sizeof(int);
	int *array;

	hipError_t cudaStatus = hipMallocManaged(&array, size);

	if (cudaStatus != hipSuccess){
		std::cerr << "hipMallocManaged failed: " << hipGetErrorString(cudaStatus) << std::endl;
    return 1;
	}

  printThread<<<4,4>>>();
  hipDeviceSynchronize();

	hipFree(array);

  return 0;
}