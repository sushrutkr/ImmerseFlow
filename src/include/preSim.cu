#include "hip/hip_runtime.h"
#include "../header/preSim.cuh"
#include "../header/postSim.cuh"
#include "../header/globalVariables.cuh"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <hip/hip_runtime.h>

#define idx(i, j, nx) ((i) + (j) * (nx))



template <unsigned int blockSize>
__device__ void warpReduce(volatile REALTYPE* sdata, unsigned int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

template <unsigned int blockSize>
__global__ void reduce6(REALTYPE* g_idata, REALTYPE* g_odata, unsigned int n) {
    extern __shared__ REALTYPE sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockSize * 2) + tid;
    unsigned int gridSize = blockSize * 2 * gridDim.x;

    sdata[tid] = 0;
    while (i  < n) {
        if (i + blockSize < n)
        {
            sdata[tid] += g_idata[i] + g_idata[i + blockSize]; i += gridSize;
        }
        else
        {
            sdata[tid] += g_idata[i]; i += gridSize;
        }
    }

    __syncthreads();

    if (blockSize >= 1024) { if (tid < 512) { sdata[tid] += sdata[tid + 512]; } __syncthreads(); }
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
    if (tid < 32) warpReduce<blockSize>(sdata, tid);
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
    
}


__global__ void initializeKernel(int nx, int ny, CFDData deviceData) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int nGrid = blockDim.x * gridDim.x;

    while (id < nx*ny)
    {
        if (id < nx * ny)
        {
            deviceData.u.velc[id] = 0.0;
            deviceData.v.velc[id] = 0.0;
            deviceData.u.velInter[id] = 0.0;
            deviceData.v.velInter[id] = 0.0;
            deviceData.p[id] = 0.0;
            
        }

        
        id = id + nGrid;
    }

    id = blockIdx.x * blockDim.x + threadIdx.x;
    while (id < (nx - 2) * ny)
    {
        if (id < (nx - 2) * ny)
        {
            deviceData.v.velf[id] = 0.0;
        }
        id = id + nGrid;
    }

    id = blockIdx.x * blockDim.x + threadIdx.x;
    while (id < nx * (ny - 2))
    {
        if (id < nx * (ny - 2))
        {
            deviceData.u.velf[id] = 0.0;
        }
        id = id + nGrid;
    }
}

__global__ void printKernel(int nx, int ny, CFDData deviceData) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int nGrid = blockDim.x * gridDim.x;
    
    while (id < nx * ny)
    {
       printf("u[%d,%d]: %f, v[%d,%d]: %f, p[%d,%d]: %f\n", id/nx, id % nx, deviceData.u.velc[id], id / nx, id % nx, deviceData.v.velc[id], id / nx, id % nx, deviceData.p[id]);
       id = id + nGrid;
    }
}

__global__ void iBlankComputeKernel(int nx, int ny, Grid gridData, IBM ibm) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int nGrid = blockDim.x * gridDim.x;

    while (id < nx * ny)
    {
        int i = id % nx;
        int j = id / nx;
        // Access x and y from gridData directly
        REALTYPE x = gridData.xc[i];
        REALTYPE y = gridData.yc[j];

        // Calculate distance from center (3, 2.5)
        REALTYPE distance = sqrtf(powf((x - 3.0f), 2) + powf((y - 2.5f), 2));

        // Check if within radius of 0.5
        if (distance <= 0.5) {
            ibm.iBlank[id] = 1.0; // Assuming linear indexing
        } else {
            ibm.iBlank[id] = 0.0;
        }
        id = id + nGrid;
    }
}

__global__ void printGridDataKernel(REALTYPE *xf, int nNodes) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int nGrid = blockDim.x * gridDim.x;
    if (id < nNodes) {
        printf("gridData[%d] = %f\n", id, xf[id]);
        id = id + nGrid;
    }
}

__global__ void jacobiIteration(int nx, int ny, Grid gridData, coeffPPE coeff, REALTYPE* p, REALTYPE* p_new) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int nGrid = blockDim.x * gridDim.x;

    while (id < nx * ny) {
        int i = id % nx;
        int j = id / nx;

        p_new[id] = p[id];

        if (i > 0 && i < nx-1 && j > 0 && j < ny-1) {
            p_new[id] = - (p[(i+1) + j * nx] * coeff.coeff_dx2_p1[id] +
                           p[(i-1) + j * nx] * coeff.coeff_dx2_m1[id] +
                           p[i + (j+1) * nx] * coeff.coeff_dy2_p1[id] + 
                           p[i + (j-1) * nx] * coeff.coeff_dy2_m1[id]) / coeff.coeff_ppe[id];
        }
        id += nGrid;
    }
}

__global__ void Compute_Residual(int nx, int ny, Grid gridData, coeffPPE coeff, REALTYPE* p, REALTYPE* Residual) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int nGrid = blockDim.x * gridDim.x;

    while (id < nx * ny) {
        int i = id % nx;
        int j = id / nx;

        if (i > 0 && i < nx - 1 && j > 0 && j < ny - 1) {
           Residual[id] = p[id] * coeff.coeff_ppe[id] + 
                p[(i + 1) + j * nx] * coeff.coeff_dx2_p1[id] +
                p[(i - 1) + j * nx] * coeff.coeff_dx2_m1[id] +
                p[i + (j + 1) * nx] * coeff.coeff_dy2_p1[id] +
                p[i + (j - 1) * nx] * coeff.coeff_dy2_m1[id] ;
        }
        id += nGrid;
    }
}

__global__ void set_pressure_BC(int nx, int ny, REALTYPE *p){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int nGrid = blockDim.x * gridDim.x;
    
    while (id < nx *ny){
        int i = id % nx;
        int j = id / nx;

        if (i==0){
            p[id] = 100.0;
        }

        if (j==0){
            p[id] = 100.0;
        }
        id += nGrid;
    }
}

__global__ void calculatePPECoefficients(int nx, int ny, Grid gridData, coeffPPE coeff) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int nGrid = blockDim.x * gridDim.x;

    while (id < nx * ny) {
        int i = id % nx;
        int j = id / nx;

        coeff.coeff_ppe[id] = 1;
        coeff.coeff_dx2_m1[id] = 1;
        coeff.coeff_dx2_p1[id] = 1;
        coeff.coeff_dy2_m1[id] = 1;
        coeff.coeff_dy2_p1[id] = 1;

        if (i > 0 && i < nx-1 && j > 0 && j < ny-1) {
            REALTYPE dx_i = gridData.dx[idx(i,j,nx)];
            REALTYPE dx_ip1 = gridData.dx[(i+1) + j * nx];
            REALTYPE dx_im1 = gridData.dx[(i-1) + j * nx];
            REALTYPE dy_j = gridData.dy[idx(i,j,nx)];
            REALTYPE dy_jp1 = gridData.dy[i + (j+1) * nx];
            REALTYPE dy_jm1 = gridData.dy[i + (j-1) * nx];

            coeff.coeff_ppe[id] = -1 * (((2 / (dx_i * (dx_i + dx_ip1))) + (2 / (dx_i * (dx_i + dx_im1)))) +
                                        ((2 / (dy_j * (dy_j + dy_jp1))) + (2 / (dy_j * (dy_j + dy_jm1)))));

            coeff.coeff_dx2_m1[id] = (2 / (dx_i * (dx_i + dx_im1)));
            coeff.coeff_dx2_p1[id] = (2 / (dx_i * (dx_i + dx_ip1)));
            coeff.coeff_dy2_m1[id] = (2 / (dy_j * (dy_j + dy_jm1)));
            coeff.coeff_dy2_p1[id] = (2 / (dy_j * (dy_j + dy_jp1)));
        }

        id += nGrid;
    }
}

void ImmerseFlow::allocation() {
    CHECK_CUDA_ERROR(hipMalloc((void**)&Data.u.velc, sizeof(REALTYPE) * Input.nx * Input.ny));
    CHECK_CUDA_ERROR(hipMalloc((void**)&Data.v.velc, sizeof(REALTYPE) * Input.nx * Input.ny));
    CHECK_CUDA_ERROR(hipMalloc((void**)&Data.u.velInter, sizeof(REALTYPE) * Input.nx * Input.ny));
    CHECK_CUDA_ERROR(hipMalloc((void**)&Data.v.velInter, sizeof(REALTYPE) * Input.nx * Input.ny));
    CHECK_CUDA_ERROR(hipMalloc((void**)&Data.u.velf, sizeof(REALTYPE) * Input.nx * (Input.ny-2)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&Data.v.velf, sizeof(REALTYPE) * (Input.nx-2) * Input.ny));
    CHECK_CUDA_ERROR(hipMallocManaged((void**)&Data.p, sizeof(REALTYPE) * Input.nx * Input.ny));
    CHECK_CUDA_ERROR(hipMallocManaged((void**)&gridData.xc, sizeof(REALTYPE) * Input.nx));
    CHECK_CUDA_ERROR(hipMalloc((void**)&gridData.yc, sizeof(REALTYPE) * Input.ny));
    CHECK_CUDA_ERROR(hipMalloc((void**)&gridData.xf, sizeof(REALTYPE) * Input.nxf));
    CHECK_CUDA_ERROR(hipMalloc((void**)&gridData.yf, sizeof(REALTYPE) * Input.nyf));
    CHECK_CUDA_ERROR(hipMalloc((void**)&gridData.dx, sizeof(REALTYPE) * Input.nx * Input.ny));
    CHECK_CUDA_ERROR(hipMalloc((void**)&gridData.dy, sizeof(REALTYPE) * Input.nx * Input.ny));
    CHECK_CUDA_ERROR(hipMalloc((void**)&ibm.iBlank, sizeof(REALTYPE) * Input.nx * Input.ny));
}

void ImmerseFlow::freeAllocation() {
    // Free allocated memory
    CHECK_CUDA_ERROR(hipFree(Data.u.velc));
    CHECK_CUDA_ERROR(hipFree(Data.v.velc));
    CHECK_CUDA_ERROR(hipFree(Data.u.velInter));
    CHECK_CUDA_ERROR(hipFree(Data.v.velInter));
    CHECK_CUDA_ERROR(hipFree(Data.u.velf));
    CHECK_CUDA_ERROR(hipFree(Data.v.velf));
    CHECK_CUDA_ERROR(hipFree(Data.p));
    CHECK_CUDA_ERROR(hipFree(gridData.xc));
    CHECK_CUDA_ERROR(hipFree(gridData.yc));
    CHECK_CUDA_ERROR(hipFree(gridData.xf));
    CHECK_CUDA_ERROR(hipFree(gridData.yf));
    CHECK_CUDA_ERROR(hipFree(ibm.iBlank));
}

void copyDataToHost(int nx, int ny, const Grid& gridData, const IBM& ibm, REALTYPE* host_x, REALTYPE* host_y, REALTYPE* host_iBlank) {
    CHECK_CUDA_ERROR(hipMemcpy(host_x, gridData.xc, sizeof(REALTYPE) * nx, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(host_y, gridData.yc, sizeof(REALTYPE) * ny, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(host_iBlank, ibm.iBlank, sizeof(REALTYPE) * nx * ny, hipMemcpyDeviceToHost));
}

void ImmerseFlow::CUDAQuery() {
    hipDeviceProp_t prop;    
    hipGetDeviceProperties(&prop, 0);
    CUDAData.threadsPerBlock = prop.maxThreadsPerBlock;
    //Do we need to change this for cell face case or just waste some cores
    CUDAData.blocksPerGrid = (Input.nx * Input.ny + CUDAData.threadsPerBlock - 1) / CUDAData.threadsPerBlock;
    printf("Maximum number of threads = %d\n", prop.maxThreadsPerBlock);
    printf("Maximum number of blocks = %d\n", prop.maxThreadsDim[0]);
}

void ImmerseFlow:: initializeData() {

    // Initialize kernel
    initializeKernel<<<CUDAData.blocksPerGrid, CUDAData.threadsPerBlock>>>(Input.nx, Input.ny, Data);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // Initialize iBlank to zero
    CHECK_CUDA_ERROR(hipMemset(ibm.iBlank, 0, sizeof(REALTYPE) * Input.nx * Input.ny));
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // Compute iBlank kernel
    iBlankComputeKernel<<<CUDAData.blocksPerGrid, CUDAData.threadsPerBlock >>>(Input.nx, Input.ny, gridData, ibm);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    saveDataToFile(Input.nx, Input.ny, gridData.xc, gridData.yc, ibm.iBlank, "../results/final_results.dat");
    
    // // Allocate host memory
    // REALTYPE* host_x = (REALTYPE*)malloc(sizeof(REALTYPE) * Input.nx);
    // REALTYPE* host_y = (REALTYPE*)malloc(sizeof(REALTYPE) * Input.ny);
    // REALTYPE* host_iBlank = (REALTYPE*)malloc(sizeof(REALTYPE) * Input.nx * Input.ny);

    // // Check allocation
    // if (host_x == nullptr || host_y == nullptr || host_iBlank == nullptr) {
    //     std::cerr << "Host memory allocation failed" << std::endl;
    //     return;
    // }

    // // Copy data from device to host
    // copyDataToHost(Input.nx, Input.ny, gridData, ibm, host_x, host_y, host_iBlank);

    // // Write results to file
    // write_results_to_file(host_x, host_y, host_iBlank, Input.nx, Input.ny, "../results/final_results.dat");

    // // Free host memory
    // free(host_x);
    // free(host_y);
    // free(host_iBlank);
}

void ImmerseFlow::readGridData() {
    REALTYPE *xf;
    REALTYPE *yf;
    int id, idy;
    std::ifstream infile;

    xf = (REALTYPE*)malloc(Input.nxf * sizeof(REALTYPE));
    yf = (REALTYPE*)malloc(Input.nyf * sizeof(REALTYPE));

    REALTYPE *x_centers = (REALTYPE*)malloc(Input.nx * sizeof(REALTYPE));
    REALTYPE *y_centers = (REALTYPE*)malloc(Input.ny * sizeof(REALTYPE));

    // Check if memory allocation was successful
    if (xf == nullptr || yf == nullptr) {
        std::cerr << "Memory allocation failed" << std::endl;
        exit(1);
    }

    // Check if memory allocation was successful
    if (x_centers == nullptr || y_centers == nullptr) {
        std::cerr << "Memory allocation failed" << std::endl;
        exit(1);
    }

    // Read values from xgrid.dat
    infile.open("../inputs/xgrid.dat2");
    if (!infile) {
        std::cerr << "Error opening xgrid.dat" << std::endl;
        free(xf);
        free(yf);
        exit(1);
    }
    for (int i = 0; i < Input.nxf; i++) {
        infile >> id >> xf[i];
    }
    infile.close();

    // Read values from ygrid.dat
    infile.open("../inputs/ygrid.dat2");
    if (!infile) {
        std::cerr << "Error opening ygrid.dat" << std::endl;
        free(xf);
        free(yf);
        exit(1);
    }
    for (int i = 0; i < Input.nyf; i++) {
        infile >> idy >> yf[i];
    }
    infile.close();

    // Calculate x centers
    for (int i = 1; i < Input.nx-1; i++) {
        x_centers[i] = (xf[i-1] + xf[i]) / 2.0;
    }

    // Calculate y centers
    for (int i = 1; i < Input.ny-1; i++) {
        y_centers[i] = (yf[i-1] + yf[i]) / 2.0;
    }

    // Handle boundary conditions
    x_centers[0] = -1 * x_centers[1];
    y_centers[0] = -1 * y_centers[1];
    x_centers[Input.nx - 1] = xf[Input.nxf - 1] + (xf[Input.nxf - 1] - x_centers[Input.nx - 2]);
    y_centers[Input.ny - 1] = yf[Input.nyf - 1] + (yf[Input.nyf - 1] - y_centers[Input.ny - 2]);

    // for (int i = 0; i < Input.nxf; ++i) {
    //     std::cout << "x[" << i << "] = " << xf[i] << std::endl;
    // }


    // Compute Grid Spacing
    // Allocate memory for dx and dy
    REALTYPE *dx = (REALTYPE*)malloc(Input.nx * Input.ny * sizeof(REALTYPE));
    REALTYPE *dy = (REALTYPE*)malloc(Input.nx * Input.ny * sizeof(REALTYPE));

    if (dx == nullptr || dy == nullptr) {
        std::cerr << "Memory allocation failed" << std::endl;
        free(xf);
        free(yf);
        free(x_centers);
        free(y_centers);
        exit(1);
    }

    // Compute grid spacings
    for (int j = 1; j < Input.ny-1; j++) {
        for (int i = 1; i < Input.nx-1; i++) {
            dx[i + j * Input.nx] = xf[i] - xf[i-1];
            dy[i + j * Input.nx] = yf[j] - yf[j-1];  // Corrected indexing
        }
    }

    // Handle boundary conditions for dx and dy
    for (int j = 0; j < Input.ny; j++) {
        dx[idx(0, j, Input.nx)] = dx[idx(1, j, Input.nx)];
        dx[idx(Input.nx-1,j,Input.nx)] = dx[idx(Input.nx-2,j,Input.nx)];
    }

    for (int i = 0; i < Input.nx; i++) {
        dx[idx(i, 0, Input.nx)] = dx[idx(i, 1, Input.nx)];
        dx[idx(i, (Input.ny-1), Input.nx)] = dx[idx(i, (Input.ny-2), Input.nx)];
    }
    
    for (int j = 0; j < Input.ny; j++) {
        dy[idx(0, j, Input.nx)] = dy[idx(1, j, Input.nx)];
        dy[idx(Input.nx-1,j,Input.nx)] = dy[idx(Input.nx-2,j,Input.nx)];
    }

    for (int i = 0; i < Input.nx; i++) {
        dy[idx(i, 0, Input.nx)] = dy[idx(i, 1, Input.nx)];
        dy[idx(i, (Input.ny-1), Input.nx)] = dy[idx(i, (Input.ny-2), Input.nx)];
    }

    // Copy data from CPU to GPU
    CHECK_CUDA_ERROR(hipMemcpy(gridData.xf, xf, Input.nxf * sizeof(REALTYPE), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(gridData.yf, yf, Input.nyf * sizeof(REALTYPE), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(gridData.xc, x_centers, Input.nx * sizeof(REALTYPE), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(gridData.yc, y_centers, Input.ny * sizeof(REALTYPE), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(gridData.dx, dx, Input.nx * Input.ny * sizeof(REALTYPE), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(gridData.dy, dy, Input.nx * Input.ny * sizeof(REALTYPE), hipMemcpyHostToDevice));

    // Free CPU memory
    free(xf);
    free(yf);
    free(x_centers);
    free(y_centers);
    free(dx);
    free(dy);
}

void ImmerseFlow::PPESolver() {

    REALTYPE* pTemp, * pResidue;
    
    coeffPPE coeff;
    
    CHECK_CUDA_ERROR(hipMallocManaged(&pTemp, sizeof(REALTYPE) * Input.nx * Input.ny));
    CHECK_CUDA_ERROR(hipMallocManaged(&pResidue, sizeof(REALTYPE) * Input.nx * Input.ny));
    CHECK_CUDA_ERROR(hipMallocManaged(&coeff.coeff_ppe, Input.nx * Input.ny * sizeof(REALTYPE)));
    CHECK_CUDA_ERROR(hipMallocManaged(&coeff.coeff_dx2_p1, Input.nx * Input.ny * sizeof(REALTYPE)));
    CHECK_CUDA_ERROR(hipMallocManaged(&coeff.coeff_dx2_m1, Input.nx * Input.ny * sizeof(REALTYPE)));
    CHECK_CUDA_ERROR(hipMallocManaged(&coeff.coeff_dy2_p1, Input.nx * Input.ny * sizeof(REALTYPE)));
    CHECK_CUDA_ERROR(hipMallocManaged(&coeff.coeff_dy2_m1, Input.nx * Input.ny * sizeof(REALTYPE)));

    //Compute Coefficient Matrix
    calculatePPECoefficients<<<CUDAData.blocksPerGrid, CUDAData.threadsPerBlock>>>(Input.nx, Input.ny, gridData, coeff);

    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    //Set Boundary Conditions    
    set_pressure_BC<<<CUDAData.blocksPerGrid, CUDAData.threadsPerBlock>>>(Input.nx, Input.ny, Data.p);    
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());      
    
    //Setting arbitrary large number to start iterations
    REALTYPE Residual = 1.0;

    int iter=0;
    while(Residual > pow(10.0,-6.0) && iter < Input.PPE_itermax) {
    //for (int iter = 0; iter < Input.PPE_itermax; iter++) {
        jacobiIteration<<<CUDAData.blocksPerGrid, CUDAData.threadsPerBlock>>>(Input.nx, Input.ny, gridData, coeff, Data.p, pTemp);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
       
        // Swap pointers
        REALTYPE *temp = Data.p;
        Data.p = pTemp;
        pTemp = temp;

        Compute_Residual << <CUDAData.blocksPerGrid, CUDAData.threadsPerBlock >> > (Input.nx, Input.ny, gridData, coeff, Data.p, pResidue);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        Reduction(pResidue, &Residual);
        
        iter += 1;
        // printf("iter = %d %f\n", iter, Residual);
    }
    
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    //Set Boundary Conditions
    set_pressure_BC<<<CUDAData.blocksPerGrid, CUDAData.threadsPerBlock>>>(Input.nx, Input.ny, Data.p);

    saveDataToFile(Input.nx, Input.ny, gridData.xc, gridData.yc, Data.p, "../results/p.dat");

    CHECK_CUDA_ERROR(hipFree(pTemp));
    CHECK_CUDA_ERROR(hipFree(coeff.coeff_ppe));
    CHECK_CUDA_ERROR(hipFree(coeff.coeff_dx2_p1));
    CHECK_CUDA_ERROR(hipFree(coeff.coeff_dx2_m1));
    CHECK_CUDA_ERROR(hipFree(coeff.coeff_dy2_p1));
    CHECK_CUDA_ERROR(hipFree(coeff.coeff_dy2_m1));
}

void ImmerseFlow::Reduction(REALTYPE *input, REALTYPE* Residual)
{
    REALTYPE* g_odata, * g_odata2;

    CHECK_CUDA_ERROR(hipMalloc((void**)&g_odata, sizeof(REALTYPE) * CUDAData.threadsPerBlock));
    CHECK_CUDA_ERROR(hipMalloc((void**)&g_odata2, sizeof(REALTYPE) * CUDAData.threadsPerBlock));
    CHECK_LAST_CUDA_ERROR();

    // Calculate Error
    switch (CUDAData.threadsPerBlock)
    {
    case 1024:
        reduce6<1024> << < CUDAData.blocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (input, g_odata, Input.nx * Input.ny); break;
    case 512:
        reduce6<512> << < CUDAData.blocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (input, g_odata, Input.nx * Input.ny); break;
    case 256:
        reduce6<256> << < CUDAData.blocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (input, g_odata, Input.nx * Input.ny); break;
    case 128:
        reduce6<128> << < CUDAData.blocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (input, g_odata, Input.nx * Input.ny); break;
    case 64:
        reduce6< 64> << < CUDAData.blocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (input, g_odata, Input.nx * Input.ny); break;
    case 32:
        reduce6< 32> << < CUDAData.blocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (input, g_odata, Input.nx * Input.ny); break;
    case 16:
        reduce6< 16> << < CUDAData.blocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (input, g_odata, Input.nx * Input.ny); break;
    case 8:
        reduce6< 8> << < CUDAData.blocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (input, g_odata, Input.nx * Input.ny); break;
    case 4:
        reduce6< 4> << < CUDAData.blocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (input, g_odata, Input.nx * Input.ny); break;
    case 2:
        reduce6< 2> << < CUDAData.blocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (input, g_odata, Input.nx * Input.ny); break;
    case 1:
        reduce6< 1> << < CUDAData.blocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (input, g_odata, Input.nx * Input.ny); break;
    }

    int BlocksPerGrid = 1;


    switch (CUDAData.threadsPerBlock)
    {
    case 1024:
        reduce6<1024> << < BlocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata2, CUDAData.threadsPerBlock); break;
    case 512:
        reduce6<512> << < BlocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata2, CUDAData.threadsPerBlock); break;
    case 256:
        reduce6<256> << < BlocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata2, CUDAData.threadsPerBlock); break;
    case 128:
        reduce6<128> << < BlocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata2, CUDAData.threadsPerBlock); break;
    case 64:
        reduce6< 64> << < BlocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata2, CUDAData.threadsPerBlock); break;
    case 32:
        reduce6< 32> << < BlocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata2, CUDAData.threadsPerBlock); break;
    case 16:
        reduce6< 16> << < BlocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata2, CUDAData.threadsPerBlock); break;
    case 8:
        reduce6< 8> << < BlocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata2, CUDAData.threadsPerBlock); break;
    case 4:
        reduce6< 4> << < BlocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata2, CUDAData.threadsPerBlock); break;
    case 2:
        reduce6< 2> << < BlocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata2, CUDAData.threadsPerBlock); break;
    case 1:
        reduce6< 1> << < BlocksPerGrid, CUDAData.threadsPerBlock, CUDAData.threadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata2, CUDAData.threadsPerBlock); break;
    }

    CHECK_CUDA_ERROR(hipMemcpy(Residual, g_odata2, sizeof(REALTYPE) * 1, hipMemcpyDeviceToHost));
}