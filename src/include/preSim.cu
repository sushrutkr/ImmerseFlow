#include "hip/hip_runtime.h"
#include "../header/preSim.cuh"
#include "../header/postSim.cuh"
#include "../header/globalVariables.cuh"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <hip/hip_runtime.h>

template <unsigned int blockSize>
__device__ void warpReduce(volatile REALTYPE* sdata, unsigned int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

template <unsigned int blockSize>
__global__ void reduce6(REALTYPE* g_idata, REALTYPE* g_odata, unsigned int n) {
    extern __shared__ REALTYPE sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockSize * 2) + tid;
    unsigned int gridSize = blockSize * 2 * gridDim.x;

    sdata[tid] = 0;
    while (i  < n) {
        if (i + blockSize < n)
        {
            sdata[tid] += g_idata[i] + g_idata[i + blockSize]; i += gridSize;
        }
        else
        {
            sdata[tid] += g_idata[i]; i += gridSize;
        }
    }

    __syncthreads();

    if (blockSize >= 1024) { if (tid < 512) { sdata[tid] += sdata[tid + 512]; } __syncthreads(); }
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
    if (tid < 32) warpReduce<blockSize>(sdata, tid);
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
    
}


__global__ void initializeKernel(int nx, int ny, CFDData deviceData) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int nGrid = blockDim.x * gridDim.x;

    while (idx < nx*ny)
    {
        if (idx < nx * ny)
        {
            deviceData.u.velc[idx] = 0.0;
            deviceData.v.velc[idx] = 0.0;
            deviceData.u.velInter[idx] = 0.0;
            deviceData.v.velInter[idx] = 0.0;
            deviceData.u.velf[idx] = 0.0;
            deviceData.v.velf[idx] = 0.0;
            deviceData.p[idx] = 0.0;
            
        }

        
        idx = idx + nGrid;
    }

    idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < (nx - 2) * ny)
    {
        if (idx < (nx - 2) * ny)
        {
            deviceData.v.velf[idx] = 0.0;
        }
        idx = idx + nGrid;
    }

    idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < nx * (ny - 2))
    {
        if (idx < nx * (ny - 2))
        {
            deviceData.u.velf[idx] = 0.0;
        }
        idx = idx + nGrid;
    }
}

__global__ void printKernel(int nx, int ny, CFDData deviceData) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int nGrid = blockDim.x * gridDim.x;
    
    while (idx < nx * ny)
    {
       printf("u[%d,%d]: %f, v[%d,%d]: %f, p[%d,%d]: %f\n", idx/nx, idx % nx, deviceData.u.velc[idx], idx / nx, idx % nx, deviceData.v.velc[idx], idx / nx, idx % nx, deviceData.p[idx]);
       idx = idx + nGrid;
    }
}

__global__ void iBlankComputeKernel(int nx, int ny, Grid gridData, IBM ibm) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int nGrid = blockDim.x * gridDim.x;

    while (idx < nx * ny)
    {
        int i = idx % nx;
        int j = idx / nx;
        // Access x and y from gridData directly
        REALTYPE x = gridData.x[i];
        REALTYPE y = gridData.y[j];

        // Calculate distance from center (3, 2.5)
        REALTYPE distance = sqrtf(powf((x - 3.0f), 2) + powf((y - 2.5f), 2));

        // Check if within radius of 0.5
        if (distance <= 0.5) {
            ibm.iBlank[idx] = 1.0; // Assuming linear indexing
        } else {
            ibm.iBlank[idx] = 0.0;
        }
        idx = idx + nGrid;
    }
}


void copyDataToHost(int nx, int ny, const Grid& gridData, const IBM& ibm, REALTYPE* host_x, REALTYPE* host_y, REALTYPE* host_iBlank) {
    // Copy data from device to host
    CHECK_CUDA_ERROR(hipMemcpy(host_x, gridData.x, sizeof(REALTYPE) * nx, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(host_y, gridData.y, sizeof(REALTYPE) * ny, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(host_iBlank, ibm.iBlank, sizeof(REALTYPE) * nx * ny, hipMemcpyDeviceToHost));
}

void ImmerseFlow::allocation() {
    CHECK_CUDA_ERROR(hipMalloc((void**)&Data.u.velc, sizeof(REALTYPE) * Input.nx * Input.ny));
    CHECK_CUDA_ERROR(hipMalloc((void**)&Data.v.velc, sizeof(REALTYPE) * Input.nx * Input.ny));
    CHECK_CUDA_ERROR(hipMalloc((void**)&Data.u.velInter, sizeof(REALTYPE) * Input.nx * Input.ny));
    CHECK_CUDA_ERROR(hipMalloc((void**)&Data.v.velInter, sizeof(REALTYPE) * Input.nx * Input.ny));
    CHECK_CUDA_ERROR(hipMalloc((void**)&Data.u.velf, sizeof(REALTYPE) * Input.nx * (Input.ny-2)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&Data.v.velf, sizeof(REALTYPE) * (Input.nx-2) * Input.ny));
    CHECK_CUDA_ERROR(hipMalloc((void**)&Data.p, sizeof(REALTYPE) * Input.nx * Input.ny));
    CHECK_CUDA_ERROR(hipMalloc((void**)&gridData.x, sizeof(REALTYPE) * Input.nx));
    CHECK_CUDA_ERROR(hipMalloc((void**)&gridData.y, sizeof(REALTYPE) * Input.ny));
    CHECK_CUDA_ERROR(hipMalloc((void**)&ibm.iBlank, sizeof(REALTYPE) * Input.nx * Input.ny));
}

void ImmerseFlow::freeAllocation() {
    // Free allocated memory
    CHECK_CUDA_ERROR(hipFree(Data.u.velc));
    CHECK_CUDA_ERROR(hipFree(Data.v.velc));
    CHECK_CUDA_ERROR(hipFree(Data.u.velInter));
    CHECK_CUDA_ERROR(hipFree(Data.v.velInter));
    CHECK_CUDA_ERROR(hipFree(Data.u.velf));
    CHECK_CUDA_ERROR(hipFree(Data.v.velf));
    CHECK_CUDA_ERROR(hipFree(Data.p));
    CHECK_CUDA_ERROR(hipFree(gridData.x));
    CHECK_CUDA_ERROR(hipFree(gridData.y));
    CHECK_CUDA_ERROR(hipFree(ibm.iBlank));
}

void ImmerseFlow::CUDAQuery() {
    hipDeviceProp_t prop;    
    hipGetDeviceProperties(&prop, 0);
    CUDAData.threadsPerBlock = prop.maxThreadsPerBlock;
    //Do we need to change this for cell face case or just waste some cores
    CUDAData.blocksPerGrid = (Input.nx * Input.ny + CUDAData.threadsPerBlock - 1) / CUDAData.threadsPerBlock;
    printf("Maximum number of threads = %d\n", prop.maxThreadsPerBlock);
    printf("Maximum number of blocks = %d\n", prop.maxThreadsDim[0]);
}

void ImmerseFlow:: initializeData() {
    // Initialize kernel
    initializeKernel<<<CUDAData.blocksPerGrid, CUDAData.threadsPerBlock>>>(Input.nx, Input.ny, Data);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // Initialize iBlank to zero
    CHECK_CUDA_ERROR(hipMemset(ibm.iBlank, 0, sizeof(REALTYPE) * Input.nx * Input.ny));
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // Compute iBlank kernel
    iBlankComputeKernel<<<CUDAData.blocksPerGrid, CUDAData.threadsPerBlock >>>(Input.nx, Input.ny, gridData, ibm);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // Allocate host memory
    REALTYPE* host_x = (REALTYPE*)malloc(sizeof(REALTYPE) * Input.nx);
    REALTYPE* host_y = (REALTYPE*)malloc(sizeof(REALTYPE) * Input.ny);
    REALTYPE* host_iBlank = (REALTYPE*)malloc(sizeof(REALTYPE) * Input.nx * Input.ny);

    // Check allocation
    if (host_x == nullptr || host_y == nullptr || host_iBlank == nullptr) {
        std::cerr << "Host memory allocation failed" << std::endl;
        return;
    }

    // Copy data from device to host
    copyDataToHost(Input.nx, Input.ny, gridData, ibm, host_x, host_y, host_iBlank);

    // Write results to file
    write_results_to_file(host_x, host_y, host_iBlank, Input.nx, Input.ny, "../results/final_results.dat");

    // Free host memory
    free(host_x);
    free(host_y);
    free(host_iBlank);


    //Test
    int BlocksPerGrid = 4;
    int ThreadsPerBlock = 1024;

    

    REALTYPE* g_idata, * g_odata;
    CHECK_CUDA_ERROR(hipMalloc((void**)&g_odata, sizeof(REALTYPE) * ThreadsPerBlock));
    CHECK_CUDA_ERROR(hipMalloc((void**)&g_idata, sizeof(REALTYPE) * Input.nx*Input.ny));
    CHECK_LAST_CUDA_ERROR();


    

    REALTYPE h_odata;
    
	switch (ThreadsPerBlock)
	{
    case 1024:
        reduce6<1024> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (Data.u.velc, g_odata, Input.nx * Input.ny); break;
	case 512:
		reduce6<512> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (Data.u.velc, g_odata, Input.nx * Input.ny); break;
	case 256:
		reduce6<256> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (Data.u.velc, g_odata, Input.nx * Input.ny); break;
	case 128:
		reduce6<128> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (Data.u.velc, g_odata, Input.nx * Input.ny); break;
	case 64:
		reduce6< 64> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (Data.u.velc, g_odata, Input.nx * Input.ny); break;
	case 32:
		reduce6< 32> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (Data.u.velc, g_odata, Input.nx * Input.ny); break;
	case 16:
		reduce6< 16> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (Data.u.velc, g_odata, Input.nx * Input.ny); break;
	case 8:
		reduce6< 8> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (Data.u.velc, g_odata, Input.nx * Input.ny); break;
	case 4:
		reduce6< 4> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (Data.u.velc, g_odata, Input.nx * Input.ny); break;
	case 2:
		reduce6< 2> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (Data.u.velc, g_odata, Input.nx * Input.ny); break;
	case 1:
		reduce6< 1> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (Data.u.velc, g_odata, Input.nx * Input.ny); break;
	}

    BlocksPerGrid = 1;


    switch (ThreadsPerBlock)
    {
    case 1024:
        reduce6<1024> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata, Input.nx * Input.ny); break;
    case 512:
        reduce6<512> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata, Input.nx * Input.ny); break;
    case 256:
        reduce6<256> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata, Input.nx * Input.ny); break;
    case 128:
        reduce6<128> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata, Input.nx * Input.ny); break;
    case 64:
        reduce6< 64> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata, Input.nx * Input.ny); break;
    case 32:
        reduce6< 32> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata, Input.nx * Input.ny); break;
    case 16:
        reduce6< 16> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata, Input.nx * Input.ny); break;
    case 8:
        reduce6< 8> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata, Input.nx * Input.ny); break;
    case 4:
        reduce6< 4> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata, Input.nx * Input.ny); break;
    case 2:
        reduce6< 2> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata, Input.nx * Input.ny); break;
    case 1:
        reduce6< 1> << < BlocksPerGrid, ThreadsPerBlock, ThreadsPerBlock * sizeof(REALTYPE) >> > (g_odata, g_odata, Input.nx * Input.ny); break;
    }

    CHECK_CUDA_ERROR(hipMemcpy(&h_odata, g_odata, sizeof(REALTYPE) * 1, hipMemcpyDeviceToHost));
    printf("%f\n", h_odata);

}




void ImmerseFlow :: readGridData() {
    REALTYPE *x;
    REALTYPE*y;
    int idx, idy;
    std::ifstream infile;

    x = (REALTYPE*)malloc(Input.nx * sizeof(REALTYPE));
    y = (REALTYPE*)malloc(Input.ny * sizeof(REALTYPE));

    // Check if memory allocation was successful
    if (x == nullptr || y == nullptr) {
        std::cerr << "Memory allocation failed" << std::endl;
        exit(1);
    }

    // Read values from xgrid.dat
    infile.open("../inputs/xgrid.dat");
    if (!infile) {
        std::cerr << "Error opening xgrid.dat" << std::endl;
        free(x);
        free(y);
        exit(1);
    }
    for (int i = 0; i < Input.nx; i++) {
        infile >> idx >> x[i];
    }
    infile.close();

    // Read values from ygrid.dat
    infile.open("../inputs/ygrid.dat");
    if (!infile) {
        std::cerr << "Error opening ygrid.dat" << std::endl;
        free(x);
        free(y);
        exit(1);
    }
    for (int i = 0; i < Input.ny; i++) {
        infile >> idy >> y[i];
    }
    infile.close();

    // Copy data from CPU to GPU
    CHECK_CUDA_ERROR(hipMemcpy(gridData.x, x, Input.nx * sizeof(REALTYPE), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(gridData.y, y, Input.ny * sizeof(REALTYPE), hipMemcpyHostToDevice));


    // Free CPU memory
    free(x);
    free(y);
}